#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>
using namespace std;

#define N 512  // Matrix size N x N

__global__ void matMulGPU(int* A, int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

void matMulCPU(int* A, int* B, int* C) {
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j) {
            int sum = 0;
            for (int k = 0; k < N; ++k)
                sum += A[i * N + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
}

void initializeMatrix(int* mat) {
	random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> dis(0, 1000 - 1);

    for (int i = 0; i < N * N; i++)
        mat[i] = dis(gen);
}

int main() {
    int size = N * N * sizeof(int);
    int *A, *B, *C, *C_cpu;
    int *d_A, *d_B, *d_C;

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);
    C_cpu = (int*)malloc(size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    initializeMatrix(A);
    initializeMatrix(B);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + 15) / 16, (N + 15) / 16);

    // GPU Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matMulGPU<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, start, stop);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    cout << "GPU Time: " << gpuTime << " ms" << endl;

    // CPU Timing
    auto cpu_start = chrono::high_resolution_clock::now();
    matMulCPU(A, B, C_cpu);
    auto cpu_end = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_time = cpu_end - cpu_start;
    cout << "CPU Time: " << cpu_time.count() * 1000 << " ms" << endl;

    // Free memory
    free(A); free(B); free(C); free(C_cpu);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
