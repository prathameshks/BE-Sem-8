#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

const int iNumberOfArrayElements = 11444777;
float *hostInput1, *hostInput2, *hostOutput, *deviceInput1, *deviceInput2, *deviceOutput;

// CUDA kernel for vector addition
__global__ void vecAddGPU(float *in1, float *in2, float *out, int len)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len)
        out[i] = in1[i] + in2[i];
}

// Fill array with random float values
void fillFloatArrayWithRandomNumbers(float *arr, int len)
{
    for (int i = 0; i < len; i++)
        arr[i] = (float)rand() / RAND_MAX;
}

// CPU vector addition
void vecAddCPU(const float *arr1, const float *arr2, float *out, int len)
{
    for (int i = 0; i < len; i++)
        out[i] = arr1[i] + arr2[i];
}

void cleanup()
{
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
}

int main()
{
    int size = iNumberOfArrayElements * sizeof(float);
    hipError_t result;

    // Allocate memory on host and device
    hostInput1 = (float *)malloc(size);
    hostInput2 = (float *)malloc(size);
    hostOutput = (float *)malloc(size);

    result = hipMalloc((void **)&deviceInput1, size);
    result = hipMalloc((void **)&deviceInput2, size);
    result = hipMalloc((void **)&deviceOutput, size);

    // Fill input arrays with random values
    fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements);
    fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements);

    // Copy input data from host to device
    hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 dimBlock(256);
    dim3 dimGrid((iNumberOfArrayElements + dimBlock.x - 1) / dimBlock.x);

    // Measure GPU execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vecAddGPU<<<dimGrid, dimBlock>>>(deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpuTime;
    hipEventElapsedTime(&gpuTime, start, stop);

    hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
    printf("Time taken for Vector Addition on GPU = %.6f ms\n", gpuTime);

    for(int i = 0; i < 10; i++)
        printf("Output[%d] = %f\n", i, hostOutput[i]); // Print first 10 results

    // Measure CPU execution time
    auto start_cpu = std::chrono::high_resolution_clock::now();
    vecAddCPU(hostInput1, hostInput2, hostOutput, iNumberOfArrayElements);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double cpuTime = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();
    printf("Time taken for Vector Addition on CPU = %.6f ms\n", cpuTime);

    cleanup();
    return 0;
}
